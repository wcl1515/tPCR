#include "hip/hip_runtime.h"

#include "mex.h"
#include "matrix.h"
#include <math.h>
#include <complex>
#include <unistd.h>
#include "fftw3.h"



#include "hipfft/hipfft.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
//#include <hipblas.h>


#include <stdio.h>
#include <string>
#include <iostream>
#include <sys/time.h>


#include <string.h>
#include <sys/time.h>

#define GET_TIME(now) { \
   struct timeval t; \
   gettimeofday(&t, NULL); \
   now = t.tv_sec + t.tv_usec/1000000.0; \
}


#define MAX_BLOCK_SZ 512

#include "tikreg_CG_kernels_d.cu"


void mexFunction( int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[] )
{
 	
    if(nrhs != 9 ) {
	printf("\nUsage:\n");
    return;
	} else if(nlhs>1) {
	printf("Too many output arguments\n");
    return;
	}

    //////////////////////////////////// fetching data from MATLAB

     int pcnt = 0;  
    const mxArray *Measurement;
    Measurement = prhs[pcnt++];       
    std::complex<double> *meas = ( std::complex<double> *) mxGetData(Measurement);
    

    
    const mxArray *Sens;
    Sens = prhs[pcnt++];       
    const int *dims_sens = mxGetDimensions(Sens);
    const int numdim_sens = mxGetNumberOfDimensions(Sens);
    std::complex<double> *sens = ( std::complex<double> *) mxGetData(Sens);
    
    int numsens;
    if (numdim_sens == 4)
        numsens = 1;
    else
        numsens = dims_sens[4];
        
    const int numdim =4;
    const int dims_sz[] = {2, dims_sens[1], dims_sens[2], dims_sens[3] };
    int w = (int)dims_sz[1];
    int h = (int)dims_sz[2];
    int d = (int)dims_sz[3];
    int totsz = w*h*d;
           
 
    const mxArray *Ipk_index;
    Ipk_index = prhs[pcnt++];       
    const int *dims_ipk = mxGetDimensions(Ipk_index);
    double *ipk_index = (double*) mxGetData(Ipk_index);

    const mxArray *Ipk_we;
    Ipk_we = prhs[pcnt++];       
    std::complex<double> *ipk_we = (std::complex<double>*) mxGetData(Ipk_we);
  
    int numP = dims_ipk[0];
    int numK = dims_ipk[1];
    
    int the_index[numP*numK];
    for(int i = 0; i < numP*numK; i++)
        the_index[i] = (int)(ipk_index[i]-1);
    
    

  
    const mxArray *Dims_pad;
    Dims_pad = prhs[pcnt++];       
    double *dims_pad_d = (double*) mxGetData(Dims_pad);
    int w_pad = (int)dims_pad_d[0];
    int h_pad = (int)dims_pad_d[1];
    int d_pad = (int)dims_pad_d[2];
    int totsz_pad  = w_pad*h_pad*d_pad;

    
    const mxArray *BPidx;
    BPidx = prhs[pcnt++];       
    int numVox= mxGetM(BPidx);
    int * bpidx = (int*) mxGetData(BPidx);
    
    const mxArray *BPmidx;
    BPmidx = prhs[pcnt++];       
   
    const mxArray *BPweight;
    BPweight = prhs[pcnt++];       
   
    
    const mxArray *Params;
    Params = prhs[pcnt++];       
    double *params = (double*) mxGetData(Params);
    int numit = (int) params[0];
    double lambda = params[1];
    int device_num = (int) params[2];
    double tol = params[3];
    int VERBOSE = (int) params[4];
    
    if (VERBOSE == 1)  
        mexPrintf("gpuDevice: %i  lambda^2: %f\n",device_num,lambda);

   /**************** Init Cuda *****************/
    
    hipError_t rv; 
    hipDevice_t dev; 
    
    if (hipCtxGetDevice(&dev) == hipSuccess)
    {
    //   hipCtx_t  pctx ;
    //   hipCtxPopCurrent(&pctx);	      
    }   
    
    mexPrintf("dev:%i\n",dev);
    
//     
//    hipSetDevice(device_num); 
// 
//    rv = hipSetDeviceFlags(hipDeviceMapHost);
//    if (rv != hipSuccess )
//    {
//       mexPrintf("Call to hipSetDeviceFlags failed\n");
//       return;
//    } 
//     
    /******** Allocate mapped tmps for dot product calc **********/
   
   int dot_threads = 128;
   int dot_blocks = w*h*d/dot_threads;
   
   double *dot_z_h,*dot_z_d;
   rv = hipHostAlloc(&dot_z_h, dot_blocks*sizeof(double), hipHostMallocMapped);

   if (rv != hipSuccess) {
      mexPrintf("Call to hipHostAlloc failed: %i\n",rv);
      return;
   } 
   hipHostGetDevicePointer(&dot_z_d, dot_z_h, 0);
   hipMemset(dot_z_d,0,dot_blocks*sizeof(double));
   
   
    /////////////////////////////////////// MALLOCs
    
    double start,finish;
     
    GET_TIME(start);
    
    hipfftDoubleComplex *tmp1,*tmp2,*tmp3, *_r, *_d, *_z , *_meas,*_sens, *_ipk_we, *tmpsens;
    int *_the_index;
    hipfftHandle            plan;
    
	plhs[0]             =  mxCreateNumericArray(numdim,dims_sz,mxGetClassID(Sens),mxREAL);
     
    std::complex<double> *res = (std::complex<double> *) mxGetData(plhs[0]);
   
    hipMalloc( (void **) &tmp1,sizeof(hipfftDoubleComplex)*totsz_pad);
    hipMalloc( (void **) &tmp2,sizeof(hipfftDoubleComplex)*totsz_pad);
    hipMalloc( (void **) &tmp3,sizeof(hipfftDoubleComplex)*totsz);
    hipMalloc( (void **) &_r,sizeof(hipfftDoubleComplex)*totsz);
    hipMalloc( (void **) &_d,sizeof(hipfftDoubleComplex)*totsz);
    hipMalloc( (void **) &_z,sizeof(hipfftDoubleComplex)*totsz);
    hipMalloc( (void **) &_meas,sizeof(hipfftDoubleComplex)*numsens*numK);

    hipMalloc( (void **) &_sens,sizeof(hipfftDoubleComplex)*numsens*totsz);
    hipMalloc( (void **) &_ipk_we,sizeof(hipfftDoubleComplex)*numP*numK);
    hipMalloc( (void **) &_the_index,sizeof(int)*numP*numK);
    hipMalloc( (void **) &tmpsens,sizeof(hipfftDoubleComplex)*numK);

    hipDeviceSynchronize();
   
    hipMemset( tmp1,0,sizeof(hipfftDoubleComplex)*totsz_pad);
    hipMemset( tmp2,0,sizeof(hipfftDoubleComplex)*totsz_pad);
    hipMemset(  tmp3,0,sizeof(hipfftDoubleComplex)*totsz);
    hipMemset(  _r,0,sizeof(hipfftDoubleComplex)*totsz);
    hipMemset(  _d,0,sizeof(hipfftDoubleComplex)*totsz);
    hipMemset(  _z,0,sizeof(hipfftDoubleComplex)*totsz);
 
     hipDeviceSynchronize();
 
  
     /************** copy data on device **********************/

     hipMemcpy( _meas, meas, sizeof(hipfftDoubleComplex)*numsens*numK, hipMemcpyHostToDevice);
     hipMemcpy( _sens, sens, sizeof(hipfftDoubleComplex)*numsens*totsz, hipMemcpyHostToDevice);
     hipMemcpy( _ipk_we, ipk_we, sizeof(hipfftDoubleComplex)*numP*numK, hipMemcpyHostToDevice);
     hipMemcpy( _the_index, the_index, sizeof(int)*numP*numK, hipMemcpyHostToDevice);
   
     hipMemcpy( ipk_we, _ipk_we, sizeof(hipfftDoubleComplex)*numP*numK, hipMemcpyDeviceToHost);
     hipMemcpy( the_index, _the_index, sizeof(int)*numP*numK, hipMemcpyDeviceToHost);
     
 
     hipDeviceSynchronize();
    
    if (VERBOSE == 1) 
        mexPrintf("numP: %i  numK: %i whd %i %i %i pad %i %i %i numsens: %i\n",numP,numK,w,h,d,w_pad,h_pad,d_pad,numsens);
            
      
    /************** copy bpidx on device **********************/
    int *_bpmidx;
    hipfftDoubleComplex *_bpweight;
    int *bpsize = (int*) malloc(sizeof(int)*numVox);
    int *bponset  = (int*) malloc(sizeof(int)*(numVox+1));
    int *_bpsize, *_bponset, *_bpidx;
    bponset[0] = 0;
    for (int j = 0; j < numVox;j++)
    {
        mxArray *Midx = mxGetCell(BPmidx,j);
        bpsize[j] = mxGetM(Midx);
        bponset[j+1] = bponset[j] + bpsize[j];
    }
    
    
    int *tmp_bpmidx;
    hipfftDoubleComplex *tmp_bpweight;
    tmp_bpmidx = (int*) malloc(sizeof(int)*bponset[numVox]);
    tmp_bpweight = (hipfftDoubleComplex*) malloc(sizeof(hipfftDoubleComplex)*bponset[numVox]);
    if (tmp_bpmidx == 0)
    {
        mexPrintf("out of mem (host)\n");
        return;
    }
    if (tmp_bpweight == 0)
    {
        mexPrintf("out of mem (host)\n");
        return;
    }
 
    
   
    for (int j = 0; j < numVox;j++)
    {
        mxArray *Midx = mxGetCell(BPmidx,j);
        mxArray *Weight = mxGetCell(BPweight,j);
        int *midx = (int*)  mxGetData(Midx);
        hipfftDoubleComplex *bpwei = (hipfftDoubleComplex*) mxGetData(Weight);
        memcpy(tmp_bpmidx + bponset[j] , midx, sizeof(int)* bpsize[j]);
        memcpy(tmp_bpweight + bponset[j] , bpwei, sizeof(hipfftDoubleComplex)* bpsize[j]);    
    }
    
       
      
    hipMalloc( (void **) &_bpmidx,sizeof(int)* bponset[numVox]);
    hipMalloc( (void **) &_bpweight,sizeof(hipfftDoubleComplex)* bponset[numVox]);
      
    hipMemcpy(_bpmidx,tmp_bpmidx,sizeof(int)*bponset[numVox], hipMemcpyHostToDevice);
    hipMemcpy(_bpweight,tmp_bpweight,sizeof(hipfftDoubleComplex)*bponset[numVox], hipMemcpyHostToDevice);
 
   
    
    free(tmp_bpmidx);
    free(tmp_bpweight);

    hipMalloc( (void **) &_bpsize,sizeof(int)* numVox);   
    hipMalloc( (void **) &_bpidx,sizeof(int)* numVox);
    hipMalloc( (void **) &_bponset,sizeof(int)* numVox+1);    
    hipMemcpy(_bpsize,bpsize,sizeof(int)* numVox, hipMemcpyHostToDevice);
    hipMemcpy(_bpidx,bpidx,sizeof(int)* numVox, hipMemcpyHostToDevice);
    hipMemcpy(_bponset,bponset,sizeof(int)* numVox+1, hipMemcpyHostToDevice);


            
    GET_TIME(finish);

    
    if (VERBOSE == 1) {
        mexPrintf("num active Vox: %i\n",numVox);    
        mexPrintf("alloc/copy time: %f\n",finish-start);
    }
        
    hipfftPlan3d(&plan, d_pad, h_pad, w_pad, HIPFFT_Z2Z) ;
        
  
     
    // thread managements 
    int vx_block = 128;
    dim3 dimBlock_vx(vx_block,1);
    dim3 dimGrid_vx (numVox/vx_block + 1,1);
 
    dim3 dimBlock_dw(d,1);
    dim3 dimGrid_dw (w,h);

    dim3 dimBlock_sq(d,1);
    dim3 dimGrid_sq (w*h,1);
  
    // for sensing 
    int sens_block = 256;
    dim3 dimBlock_se(sens_block,1);
    dim3 dimGrid_se (numK/sens_block + 1,1);

 
    
     double AA_time = 0;
     double cg_time = 0;
     
      int err;
    
   
     double normrr = 0;
     double dAAd = 0;
     double alpha = 0;
     double normrr2 = 0;
     double beta = 0;
      
    /////////////////////////////////////////////////////// init CG
    

     // we need this because first fft fails
    int _res = hipfftExecZ2Z(plan, tmp1, tmp2, HIPFFT_FORWARD) ;
    if (VERBOSE == 1)
      mexPrintf("first fft call ret: %i\n",_res);

     
    hipMemset(_r,0, sizeof(hipfftDoubleComplex)*totsz);
    hipMemset(tmp3,0, sizeof(hipfftDoubleComplex)*totsz);
    hipMemset(_z,0, sizeof(hipfftDoubleComplex)*totsz);   
    hipMemset( tmp2,0,sizeof(hipfftDoubleComplex)*totsz_pad);
   
                
    // backproject measurement -- x=A'b
    for (int i = 0; i < numsens; i++)
    { 
        hipMemset(tmp1,0, sizeof(hipfftDoubleComplex)*totsz_pad);
        backprojVX<<<dimGrid_vx,dimBlock_vx>>>(_bpidx,_bponset,_bpweight,_bpmidx,_bpsize,_meas + i*numK, tmp1,numVox);
        if (err=hipfftExecZ2Z(plan, tmp1, tmp2, HIPFFT_BACKWARD) != HIPFFT_SUCCESS)
        {
            mexPrintf("1) cufft has failed with err %i \n",err);
            return;
        }    
        downwind<<<dimGrid_dw,dimBlock_dw>>>(_r,tmp2, _sens + i*totsz, w, h, d, w_pad, h_pad, d_pad);
     }
  
     hipMemcpy( res, _r, sizeof(hipfftDoubleComplex)*totsz,hipMemcpyDeviceToHost);    
     hipMemcpy( _d, _r, sizeof(hipfftDoubleComplex)*totsz,hipMemcpyDeviceToDevice);
     
    
     normrr = Dot_wrapper(_r, _r, dot_z_d, dot_z_h, totsz, dot_blocks, dot_threads) ;
     
    // normrr = hipblasCdotc(totsz,(hipComplex*)_r,1,(hipComplex*)_r,1).x;           
     
     double normrr0 = normrr;
     if (VERBOSE == 1)
        mexPrintf("first residual: %f\n",normrr0);
   
    
     
    ////////////////////////////////////////////////////////////// start CG
    GET_TIME(start);
   
    for (int it = 0; it < numit; it++)
    {
     
        GET_TIME(start);
        scmult<<<dimGrid_sq,dimBlock_sq>>>(tmp3,_d,lambda,totsz);
        GET_TIME(finish); cg_time += finish-start;
        
        for (int i = 0; i < numsens; i++)
        {
            
            hipMemset(tmp1,0, sizeof(hipfftDoubleComplex)*totsz_pad);           
            upwind<<<dimGrid_dw,dimBlock_dw>>>(tmp1,_d, _sens + i*totsz, w, h, d, w_pad, h_pad, d_pad);            
            if (err=hipfftExecZ2Z(plan, tmp1, tmp2, HIPFFT_FORWARD) != HIPFFT_SUCCESS)
            {
                mexPrintf("2) cufft has failed with err %i \n",err);
                return;
            }
            hipMemset(tmpsens,0, sizeof(hipfftDoubleComplex)*numK);
            
            
            dosens<<<dimGrid_se,dimBlock_se>>>(tmpsens,tmp2,_ipk_we,_the_index,numP,numK);
            hipMemset(tmp1,0, sizeof(hipfftDoubleComplex)*totsz_pad);            
            
            
            backprojVX<<<dimGrid_vx,dimBlock_vx>>>(_bpidx,_bponset,_bpweight,_bpmidx,_bpsize,tmpsens, tmp1,numVox);
              
                        
            if (err=hipfftExecZ2Z(plan, tmp1, tmp2, HIPFFT_BACKWARD) != HIPFFT_SUCCESS)
            {
                mexPrintf("3) cufft has failed with err %i \n",err);
                return;
            }
            downwind<<<dimGrid_dw,dimBlock_dw>>>(tmp3,tmp2, _sens + i*totsz, w, h, d, w_pad, h_pad, d_pad);                
         
        }
        
        hipDeviceSynchronize();
       
        GET_TIME(finish); AA_time += finish-start;
               
        GET_TIME(start);    
        dAAd = Dot_wrapper(_d, tmp3, dot_z_d, dot_z_h, totsz, dot_blocks, dot_threads) ;        
        //dAAd = hipblasCdotc(totsz,(hipComplex*)_d,1,(hipComplex*)tmp3,1).x;           
     
        alpha = normrr/dAAd;        
        scpm<<<dimGrid_sq,dimBlock_sq>>>(_z,_d,alpha,totsz);
        scpm<<<dimGrid_sq,dimBlock_sq>>>(_r,tmp3,-alpha,totsz);                       	
        normrr2 = Dot_wrapper(_r, _r, dot_z_d, dot_z_h, totsz, dot_blocks, dot_threads) ;   
        //normrr2 = hipblasCdotc(totsz,(hipComplex*)_r,1,(hipComplex*)_r,1).x;     
        beta = normrr2/normrr;
        normrr = normrr2;        
        scmultplus<<<dimGrid_sq,dimBlock_sq>>>(_d,_r,beta,totsz);     
        
        hipDeviceSynchronize();
        
        GET_TIME(finish);  cg_time += finish-start;
       
        if (sqrt(normrr/normrr0) < tol)
            break;       
        
            
        if (VERBOSE == 1)  
            mexPrintf("tol: %f\n",sqrt(normrr/normrr0));

        mexPrintf("."); mexEvalString("drawnow");
    
    }
         
   
    if (VERBOSE == 1)
    {
        mexPrintf("\n");        
        mexPrintf(" AA time: %f \n",AA_time);
        mexPrintf(" cg  time: %f \n",cg_time);
    }
        
    hipMemcpy( res, _z, sizeof(hipfftDoubleComplex)*totsz,hipMemcpyDeviceToHost);
   

//    hipHostFree(dot_z_h);     
    hipFree(tmp1);
    hipFree(tmp2);
    hipFree(tmp3);
    hipFree(_r); 
    hipFree(_d);
    hipFree(_z);
    hipFree(_meas);
    hipFree(_sens);
    hipFree(_ipk_we);
    hipFree(_the_index);
    hipFree(tmpsens);
    hipFree(_bpmidx);
    hipFree(_bpweight);
    hipFree(_bpsize);
    hipFree(_bpidx);
    hipFree(_bponset);    
    
    hipfftDestroy(plan);
    free(bpsize);
    free(bponset);
 

     hipCtx_t  pctx ;
     hipCtxPopCurrent(&pctx);	
    
}













