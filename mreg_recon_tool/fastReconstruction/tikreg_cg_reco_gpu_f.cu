#include "hip/hip_runtime.h"

#include "mex.h"
#include "matrix.h"
#include <math.h>
#include <complex>
#include <unistd.h>
#include "fftw3.h"



#include "hipfft/hipfft.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include <hipblas.h>


#include <stdio.h>
#include <string>
#include <iostream>
#include <sys/time.h>


#include <string.h>
#include <sys/time.h>

#define GET_TIME(now) { \
   struct timeval t; \
   gettimeofday(&t, NULL); \
   now = t.tv_sec + t.tv_usec/1000000.0; \
}


#define MAX_BLOCK_SZ 512


#include "tikreg_CG_kernels.cu"


void mexFunction( int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[] )
{
 	
    if(nrhs != 9 ) {
	printf("\nUsage:\n");
    return;
	} else if(nlhs>1) {
	printf("Too many output arguments\n");
    return;
	}

    //////////////////////////////////// fetching data from MATLAB

     int pcnt = 0;  
    const mxArray *Measurement;
    Measurement = prhs[pcnt++];       
    std::complex<float> *meas = ( std::complex<float> *) mxGetData(Measurement);
    

    
    const mxArray *Sens;
    Sens = prhs[pcnt++];       
    const int *dims_sens = mxGetDimensions(Sens);
    const int numdim_sens = mxGetNumberOfDimensions(Sens);
    std::complex<float> *sens = ( std::complex<float> *) mxGetData(Sens);
    
    int numsens;
    if (numdim_sens == 4)
        numsens = 1;
    else
        numsens = dims_sens[4];
        
    const int numdim =4;
    const int dims_sz[] = {2, dims_sens[1], dims_sens[2], dims_sens[3] };
    int w = (int)dims_sz[1];
    int h = (int)dims_sz[2];
    int d = (int)dims_sz[3];
    int totsz = w*h*d;
           
 
    const mxArray *Ipk_index;
    Ipk_index = prhs[pcnt++];       
    const int *dims_ipk = mxGetDimensions(Ipk_index);
    float *ipk_index = (float*) mxGetData(Ipk_index);

    const mxArray *Ipk_we;
    Ipk_we = prhs[pcnt++];       
    std::complex<float> *ipk_we = (std::complex<float>*) mxGetData(Ipk_we);
  
    int numP = dims_ipk[0];
    int numK = dims_ipk[1];
    
    int the_index[numP*numK];
    for(int i = 0; i < numP*numK; i++)
        the_index[i] = (int)(ipk_index[i]-1);
    
    

  
    const mxArray *Dims_pad;
    Dims_pad = prhs[pcnt++];       
    float *dims_pad_d = (float*) mxGetData(Dims_pad);
    int w_pad = (int)dims_pad_d[0];
    int h_pad = (int)dims_pad_d[1];
    int d_pad = (int)dims_pad_d[2];
    int totsz_pad  = w_pad*h_pad*d_pad;

    
    const mxArray *BPidx;
    BPidx = prhs[pcnt++];       
    int numVox= mxGetM(BPidx);
    int * bpidx = (int*) mxGetData(BPidx);
    
    const mxArray *BPmidx;
    BPmidx = prhs[pcnt++];       
   
    const mxArray *BPweight;
    BPweight = prhs[pcnt++];       
   
    
    const mxArray *Params;
    Params = prhs[pcnt++];       
    float *params = (float*) mxGetData(Params);
    int numit = (int) params[0];
    float lambda = params[1];
    int device_num = (int) params[2];
    float tol = params[3];
    int VERBOSE = (int) params[4];
    
    if (VERBOSE == 1)  
        mexPrintf("gpuDevice: %i  lambda^2: %f\n",device_num,lambda);

   /**************** Init Cuda *****************/
    
    hipError_t rv; 
    hipDevice_t dev; 
    
    if (hipCtxGetDevice(&dev) == hipSuccess)
    {
    //   hipCtx_t  pctx ;
    //   hipCtxPopCurrent(&pctx);	      
    }   
    
    mexPrintf("dev:%i\n",dev);
    
//     
//    hipSetDevice(device_num); 
// 
//    rv = hipSetDeviceFlags(hipDeviceMapHost);
//    if (rv != hipSuccess )
//    {
//       mexPrintf("Call to hipSetDeviceFlags failed\n");
//       return;
//    } 
//     
//     /******** Allocate mapped tmps for dot product calc **********/
//    
//    int dot_threads = 128;
//    int dot_blocks = w*h*d/dot_threads;
//    
//    float *dot_z_h,*dot_z_d;
//    rv = hipHostAlloc(&dot_z_h, dot_blocks*sizeof(float), hipHostMallocMapped);
// 
//    if (rv != hipSuccess) {
//       mexPrintf("Call to hipHostAlloc failed: %i\n",rv);
//       return;
//    } 
//    hipHostGetDevicePointer(&dot_z_d, dot_z_h, 0);
//    hipMemset(dot_z_d,0,dot_blocks*sizeof(float));
//    
   
    /////////////////////////////////////// MALLOCs
    
    double start,finish;
     
    GET_TIME(start);
    
    hipfftComplex *tmp1,*tmp2,*tmp3, *_r, *_d, *_z , *_meas,*_sens, *_ipk_we, *tmpsens;
    int *_the_index;
    hipfftHandle            plan;
    
	plhs[0]             =  mxCreateNumericArray(numdim,dims_sz,mxGetClassID(Sens),mxREAL);
     
    std::complex<float> *res = (std::complex<float> *) mxGetData(plhs[0]);
   
    hipMalloc( (void **) &tmp1,sizeof(hipfftComplex)*totsz_pad);
    hipMalloc( (void **) &tmp2,sizeof(hipfftComplex)*totsz_pad);
    hipMalloc( (void **) &tmp3,sizeof(hipfftComplex)*totsz);
    hipMalloc( (void **) &_r,sizeof(hipfftComplex)*totsz);
    hipMalloc( (void **) &_d,sizeof(hipfftComplex)*totsz);
    hipMalloc( (void **) &_z,sizeof(hipfftComplex)*totsz);
    hipMalloc( (void **) &_meas,sizeof(hipfftComplex)*numsens*numK);

    hipMalloc( (void **) &_sens,sizeof(hipfftComplex)*numsens*totsz);
    hipMalloc( (void **) &_ipk_we,sizeof(hipfftComplex)*numP*numK);
    hipMalloc( (void **) &_the_index,sizeof(int)*numP*numK);
    hipMalloc( (void **) &tmpsens,sizeof(hipfftComplex)*numK);

    hipDeviceSynchronize();
   
    hipMemset( tmp1,0,sizeof(hipfftComplex)*totsz_pad);
    hipMemset( tmp2,0,sizeof(hipfftComplex)*totsz_pad);
    hipMemset(  tmp3,0,sizeof(hipfftComplex)*totsz);
    hipMemset(  _r,0,sizeof(hipfftComplex)*totsz);
    hipMemset(  _d,0,sizeof(hipfftComplex)*totsz);
    hipMemset(  _z,0,sizeof(hipfftComplex)*totsz);
 
     hipDeviceSynchronize();
 
  
     /************** copy data on device **********************/

     hipMemcpy( _meas, meas, sizeof(hipfftComplex)*numsens*numK, hipMemcpyHostToDevice);
     hipMemcpy( _sens, sens, sizeof(hipfftComplex)*numsens*totsz, hipMemcpyHostToDevice);
     hipMemcpy( _ipk_we, ipk_we, sizeof(hipfftComplex)*numP*numK, hipMemcpyHostToDevice);
     hipMemcpy( _the_index, the_index, sizeof(int)*numP*numK, hipMemcpyHostToDevice);
   
     hipMemcpy( ipk_we, _ipk_we, sizeof(hipfftComplex)*numP*numK, hipMemcpyDeviceToHost);
     hipMemcpy( the_index, _the_index, sizeof(int)*numP*numK, hipMemcpyDeviceToHost);
     
 
     hipDeviceSynchronize();
    
    if (VERBOSE == 1) 
        mexPrintf("numP: %i  numK: %i whd %i %i %i pad %i %i %i numsens: %i\n",numP,numK,w,h,d,w_pad,h_pad,d_pad,numsens);
            
      
    /************** copy bpidx on device **********************/
    int *_bpmidx;
    hipfftComplex *_bpweight;
    int *bpsize = (int*) malloc(sizeof(int)*numVox);
    int *bponset  = (int*) malloc(sizeof(int)*(numVox+1));
    int *_bpsize, *_bponset, *_bpidx;
    bponset[0] = 0;
    for (int j = 0; j < numVox;j++)
    {
        mxArray *Midx = mxGetCell(BPmidx,j);
        bpsize[j] = mxGetM(Midx);
        bponset[j+1] = bponset[j] + bpsize[j];
    }
    
    int *tmp_bpmidx;
    hipfftComplex *tmp_bpweight;
    tmp_bpmidx = (int*) malloc(sizeof(int)*bponset[numVox]);
    tmp_bpweight = (hipfftComplex*) malloc(sizeof(hipfftComplex)*bponset[numVox]);
    if (tmp_bpmidx == 0)
    {
        mexPrintf("out of mem (host)\n");
        return;
    }
    if (tmp_bpweight == 0)
    {
        mexPrintf("out of mem (host)\n");
        return;
    }
    
    for (int j = 0; j < numVox;j++)
    {
        mxArray *Midx = mxGetCell(BPmidx,j);
        mxArray *Weight = mxGetCell(BPweight,j);
        int *midx = (int*)  mxGetData(Midx);
        hipfftComplex *bpwei = (hipfftComplex*) mxGetData(Weight);
        memcpy(tmp_bpmidx + bponset[j] , midx, sizeof(int)* bpsize[j]);
        memcpy(tmp_bpweight + bponset[j] , bpwei, sizeof(hipfftComplex)* bpsize[j]);    
    }
    
    hipMalloc( (void **) &_bpmidx,sizeof(int)* bponset[numVox]);
    hipMalloc( (void **) &_bpweight,sizeof(hipfftComplex)* bponset[numVox]);
      
    hipMemcpy(_bpmidx,tmp_bpmidx,sizeof(int)*bponset[numVox], hipMemcpyHostToDevice);
    hipMemcpy(_bpweight,tmp_bpweight,sizeof(hipfftComplex)*bponset[numVox], hipMemcpyHostToDevice);
 
    free(tmp_bpmidx);
    free(tmp_bpweight);

    hipMalloc( (void **) &_bpsize,sizeof(int)* numVox);   
    hipMalloc( (void **) &_bpidx,sizeof(int)* numVox);
    hipMalloc( (void **) &_bponset,sizeof(int)* numVox+1);    
    hipMemcpy(_bpsize,bpsize,sizeof(int)* numVox, hipMemcpyHostToDevice);
    hipMemcpy(_bpidx,bpidx,sizeof(int)* numVox, hipMemcpyHostToDevice);
    hipMemcpy(_bponset,bponset,sizeof(int)* numVox+1, hipMemcpyHostToDevice);


            
    GET_TIME(finish);

    
    if (VERBOSE == 1) {
        mexPrintf("num active Vox: %i\n",numVox);    
        mexPrintf("alloc/copy time: %f\n",finish-start);
    }
        
    hipfftPlan3d(&plan, d_pad, h_pad, w_pad, HIPFFT_C2C) ;
        
  
     
    // thread managements 
    int vx_block = 128;
    dim3 dimBlock_vx(vx_block,1);
    dim3 dimGrid_vx (numVox/vx_block + 1,1);
 
    dim3 dimBlock_dw(d,1);
    dim3 dimGrid_dw (w,h);

    dim3 dimBlock_sq(d,1);
    dim3 dimGrid_sq (w*h,1);
  
    // for sensing 
    int sens_block = 256;
    dim3 dimBlock_se(sens_block,1);
    dim3 dimGrid_se (numK/sens_block + 1,1);

 
    
     double AA_time = 0;
     double cg_time = 0;
     
      int err;
    
   
     float normrr = 0;
     float dAAd = 0;
     float alpha = 0;
     float normrr2 = 0;
     float beta = 0;
      
    /////////////////////////////////////////////////////// init CG
    

     // we need this because first fft fails
     int _res = hipfftExecC2C(plan, tmp1, tmp2, HIPFFT_FORWARD);
    if (VERBOSE == 1)
      mexPrintf("first fft call ret: %i\n", _res);

     
    hipMemset(_r,0, sizeof(hipfftComplex)*totsz);
    hipMemset(tmp3,0, sizeof(hipfftComplex)*totsz);
    hipMemset(_z,0, sizeof(hipfftComplex)*totsz);   
    hipMemset( tmp2,0,sizeof(hipfftComplex)*totsz_pad);
   
                   
    // backproject measurement -- x=A'b
    for (int i = 0; i < numsens; i++)
    { 
        hipMemset(tmp1,0, sizeof(hipfftComplex)*totsz_pad);
        backprojVX<<<dimGrid_vx,dimBlock_vx>>>(_bpidx,_bponset,_bpweight,_bpmidx,_bpsize,_meas + i*numK, tmp1,numVox);
        if (err=hipfftExecC2C(plan, tmp1, tmp2, HIPFFT_BACKWARD) != HIPFFT_SUCCESS)
        {
            mexPrintf("cufft has failed with err %i \n",err);
            return;
        }    
        downwind<<<dimGrid_dw,dimBlock_dw>>>(_r,tmp2, _sens + i*totsz, w, h, d, w_pad, h_pad, d_pad);
     }
  
     hipMemcpy( res, _r, sizeof(hipfftComplex)*totsz,hipMemcpyDeviceToHost);    
     hipMemcpy( _d, _r, sizeof(hipfftComplex)*totsz,hipMemcpyDeviceToDevice);
     
    
     //normrr = Dot_wrapper(_r, _r, dot_z_d, dot_z_h, totsz, dot_blocks, dot_threads) ;
     
     normrr = hipblasCdotc(totsz,(hipComplex*)_r,1,(hipComplex*)_r,1).x;           
     
     float normrr0 = normrr;
     if (VERBOSE == 1)
        mexPrintf("first residual: %f\n",normrr0);
   
    
     
    ////////////////////////////////////////////////////////////// start CG
    GET_TIME(start);
   
    for (int it = 0; it < numit; it++)
    {
     
        GET_TIME(start);
        scmult<<<dimGrid_sq,dimBlock_sq>>>(tmp3,_d,lambda,totsz);
        GET_TIME(finish); cg_time += finish-start;
        
        for (int i = 0; i < numsens; i++)
        {
            
            hipMemset(tmp1,0, sizeof(hipfftComplex)*totsz_pad);           
            upwind<<<dimGrid_dw,dimBlock_dw>>>(tmp1,_d, _sens + i*totsz, w, h, d, w_pad, h_pad, d_pad);            
            if (err=hipfftExecC2C(plan, tmp1, tmp2, HIPFFT_FORWARD) != HIPFFT_SUCCESS)
            {
                mexPrintf("cufft has failed with err %i \n",err);
                return;
            }
            hipMemset(tmpsens,0, sizeof(hipfftComplex)*numK);
            
            
            dosens<<<dimGrid_se,dimBlock_se>>>(tmpsens,tmp2,_ipk_we,_the_index,numP,numK);
            hipMemset(tmp1,0, sizeof(hipfftComplex)*totsz_pad);            
            
            
            backprojVX<<<dimGrid_vx,dimBlock_vx>>>(_bpidx,_bponset,_bpweight,_bpmidx,_bpsize,tmpsens, tmp1,numVox);
              
                        
            if (err=hipfftExecC2C(plan, tmp1, tmp2, HIPFFT_BACKWARD) != HIPFFT_SUCCESS)
            {
                mexPrintf("cufft has failed with err %i \n",err);
                return;
            }
            downwind<<<dimGrid_dw,dimBlock_dw>>>(tmp3,tmp2, _sens + i*totsz, w, h, d, w_pad, h_pad, d_pad);                
         
        }
        
        hipDeviceSynchronize();
       
        GET_TIME(finish); AA_time += finish-start;
               
        GET_TIME(start);    
        //dAAd = Dot_wrapper(_d, tmp3, dot_z_d, dot_z_h, totsz, dot_blocks, dot_threads) ;        
        dAAd = hipblasCdotc(totsz,(hipComplex*)_d,1,(hipComplex*)tmp3,1).x;           
     
        alpha = normrr/dAAd;        
        scpm<<<dimGrid_sq,dimBlock_sq>>>(_z,_d,alpha,totsz);
        scpm<<<dimGrid_sq,dimBlock_sq>>>(_r,tmp3,-alpha,totsz);                       	
        //normrr2 = Dot_wrapper(_r, _r, dot_z_d, dot_z_h, totsz, dot_blocks, dot_threads) ;   
        normrr2 = hipblasCdotc(totsz,(hipComplex*)_r,1,(hipComplex*)_r,1).x;     
        beta = normrr2/normrr;
        normrr = normrr2;        
        scmultplus<<<dimGrid_sq,dimBlock_sq>>>(_d,_r,beta,totsz);     
        
        hipDeviceSynchronize();
        
        GET_TIME(finish);  cg_time += finish-start;
       
        if (sqrt(normrr/normrr0) < tol)
            break;       
        
            
        if (VERBOSE == 1)  
            mexPrintf("tol: %f\n",sqrt(normrr/normrr0));

        mexPrintf("."); mexEvalString("drawnow");
    
    }
         
   
    if (VERBOSE == 1)
    {
        mexPrintf("\n");        
        mexPrintf(" AA time: %f \n",AA_time);
        mexPrintf(" cg  time: %f \n",cg_time);
    }
        
    hipMemcpy( res, _z, sizeof(hipfftComplex)*totsz,hipMemcpyDeviceToHost);
   

//    hipHostFree(dot_z_h);     
    hipFree(tmp1);
    hipFree(tmp2);
    hipFree(tmp3);
    hipFree(_r); 
    hipFree(_d);
    hipFree(_z);
    hipFree(_meas);
    hipFree(_sens);
    hipFree(_ipk_we);
    hipFree(_the_index);
    hipFree(tmpsens);
    hipFree(_bpmidx);
    hipFree(_bpweight);
    hipFree(_bpsize);
    hipFree(_bpidx);
    hipFree(_bponset);    
    
    hipfftDestroy(plan);
    free(bpsize);
    free(bponset);
 

     hipCtx_t  pctx ;
     hipCtxPopCurrent(&pctx);	
    
}













