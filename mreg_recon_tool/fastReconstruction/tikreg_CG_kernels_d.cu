#include "hip/hip_runtime.h"


static __global__ void  backproj(hipfftDoubleComplex *val,
                                    hipfftDoubleComplex *tmp1,
                                     hipfftDoubleComplex *_ipk_we,
                                     int *_the_index)
{
      int p   = threadIdx.x;
      int idx = _the_index[p];
      tmp1[idx].x +=   val[0].x * _ipk_we[p].x +  val[0].y*_ipk_we[p].y;
      tmp1[idx].y += - val[0].x*_ipk_we[p].y +  val[0].y*_ipk_we[p].x ;  
}



static __global__ void  backprojWS(hipfftDoubleComplex *val,
                                    hipfftDoubleComplex *tmp1,
                                     hipfftDoubleComplex *_ipk_we,
                                     int *_the_index, int numP, int *ws_indices, int ws_size)
{
      int p = threadIdx.x;
      int k = blockIdx.x; 
      if (k < ws_size)
      {
          int j = ws_indices[k];     
          int q = p + numP*j;

          int idx = _the_index[q];
          tmp1[idx].x +=   val[j].x * _ipk_we[q].x +  val[j].y*_ipk_we[q].y;
          tmp1[idx].y += - val[j].x *_ipk_we[q].y  +  val[j].y*_ipk_we[q].x;  
      }
}
     

static __global__ void  backprojVX(int *vxIdx,
                                    int *onset,
                                    hipfftDoubleComplex *we, 
                                    int *id,
                                    int *sz, 
                                    hipfftDoubleComplex *val, hipfftDoubleComplex *tmp1, int numVox)
                                  
{
      int t = blockIdx.x*blockDim.x + threadIdx.x;
      if (t < numVox)
      {
          int idx = vxIdx[t];
          int ons = onset[t];
          int size = sz[t];
          
          for (int k = 0; k < size; k++)
          {          
              int j = id[ons+k];
              tmp1[idx].x +=   val[j].x * we[ons+k].x +  val[j].y*we[ons+k].y;
              tmp1[idx].y +=   - val[j].x * we[ons+k].y  +  val[j].y*we[ons+k].x;            
          }
      }
}




static __global__ void  dosens(hipfftDoubleComplex *val,
                                    hipfftDoubleComplex *tmp2,
                                     hipfftDoubleComplex *_ipk_we,
                                     int *_the_index,int numP, int numK)
{     
      int k = blockDim.x * blockIdx.x + threadIdx.x;
      if (k < numK)
      {
          val[k].x = 0;  val[k].y = 0;          
          for (int p = 0; p < numP; p++)
          { 
              int idx = _the_index[numP*k + p];          
              val[k].x += tmp2[idx].x*_ipk_we[numP*k + p].x - tmp2[idx].y*_ipk_we[numP*k + p].y;
              val[k].y += tmp2[idx].x*_ipk_we[numP*k + p].y + tmp2[idx].y*_ipk_we[numP*k + p].x;
          }
      }
}




static __global__ void  downwind(hipfftDoubleComplex *_r,
                                    hipfftDoubleComplex *tmp2,
                                     hipfftDoubleComplex *_sens, int w, int h, int d, int w_pad, int h_pad, int d_pad)
{
    int z = threadIdx.x;
    int y = blockIdx.y;
    int x = blockIdx.x;
    int idx_pad = z*w_pad*h_pad+y*w_pad+x;
    int idx = z*w*h + y*w + x;
    
    _r[idx].x +=  tmp2[idx_pad].x*_sens[idx].x + tmp2[idx_pad].y*_sens[idx].y;
    _r[idx].y +=  - tmp2[idx_pad].x*_sens[idx].y + tmp2[idx_pad].y*_sens[idx].x;

}

static __global__ void  upwind(hipfftDoubleComplex *_r,
                                    hipfftDoubleComplex *tmp2,
                                     hipfftDoubleComplex *_sens, int w, int h, int d, int w_pad, int h_pad, int d_pad)
{
    int z = threadIdx.x;
    int y = blockIdx.y;
    int x = blockIdx.x;
    int idx_pad = z*w_pad*h_pad+y*w_pad+x;
    int idx = z*w*h + y*w + x;
    
    _r[idx_pad].x +=  tmp2[idx].x*_sens[idx].x - tmp2[idx].y*_sens[idx].y;
    _r[idx_pad].y +=  tmp2[idx].x*_sens[idx].y + tmp2[idx].y*_sens[idx].x;

}



static __global__ void  scmult(hipfftDoubleComplex *_a,hipfftDoubleComplex *_b, double alpha, int n)
{
    int t = blockDim.x * blockIdx.x + threadIdx.x;
    if (t < n)
    {        
         _a[t].x = _b[t].x * alpha;
         _a[t].y = _b[t].y * alpha;
    }
}



static __global__ void  scmultplus(hipfftDoubleComplex *_a,hipfftDoubleComplex *_b, double alpha, int n)
{
    int t = blockDim.x * blockIdx.x + threadIdx.x;
    if (t < n)
    {        
         _a[t].x = _a[t].x * alpha + _b[t].x;
         _a[t].y = _a[t].y * alpha + _b[t].y;
    }
}



static __global__ void  scpm(hipfftDoubleComplex *_a,hipfftDoubleComplex *_b, double alpha, int n)
{
    int t = blockDim.x * blockIdx.x + threadIdx.x;
    if (t < n)
    {        
         _a[t].x += _b[t].x * alpha;
         _a[t].y += _b[t].y * alpha;
    }    
}

__global__ void Dev_dot(hipfftDoubleComplex x[], hipfftDoubleComplex y[], double z[], int n) {
   /* Use tmp to store products of vector components in each block */
   /* Can't use variable dimension here                            */
   __shared__ double tmp[MAX_BLOCK_SZ];
   int t = blockDim.x * blockIdx.x + threadIdx.x;
   int loc_t = threadIdx.x;


   if (t < n) 
   {
       tmp[loc_t] = x[t].x*y[t].x + x[t].y*y[t].y;
   }
   __syncthreads();
   
   /* This uses a tree structure to do the addtions */
   for (int stride = blockDim.x/2; stride >  0; stride /= 2) {
      if (loc_t < stride)
      {
         tmp[loc_t] += tmp[loc_t + stride];
      }
      __syncthreads();
   }

   /* Store the result from this cache block in z[blockIdx.x] */
   if (threadIdx.x == 0) {
      z[blockIdx.x] = tmp[0];
   }
}  /* Dev_dot */    

 
double Dot_wrapper(hipfftDoubleComplex x_d[], hipfftDoubleComplex y_d[], double z_d[], double z_h[],
      int n, int blocks, int threads) { 
   int i;
   double dot = 0;

   /* Invoke kernel */
   Dev_dot<<<blocks, threads>>>(x_d, y_d, z_d, n);
   hipDeviceSynchronize();

   /* Note that we don't need to copy z_d back to host */
   for (i = 0; i < blocks; i++)
   {
      dot += z_h[i];
   }
   return dot;
} 


